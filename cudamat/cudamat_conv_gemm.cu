#include "hip/hip_runtime.h"
/** Kernels for convUp, convDown, convOutp, maxpool, avgpool, maxpoolundo,
 *  avgpoolundo.
 *  These kernels are 10-20% slower than cuda-convnet2, but have no constraints
 *  on number of channels and support rectangular images and rectangular kernels.
 *  They use hipblasSgemm for convUp, convDown, convOutp.
 *  Data layout : Column-major
 *  data : (num_images, image_size_x, image_size_y, num_input_channels)
 *  filters : (num_output_channels, kernel_size_x, kernel_size_y, num_input_channels)
 */

#include "cudamat_conv_gemm.cuh"
#define getLastCudaError(msg)   __getLastCudaError (msg, __FILE__, __LINE__)
size_t free_space_ = 0;
void EstimateFreeSpace() {
  /*
  size_t total;
  hipMemGetInfo(&free_space_, &total);
  //free_space_ >>= 1;
  if (free_space_ > 1>>20) {
    free_space_ -= 1 << 20; // Just remove 1 MB. This seems to work!
  }
  */
  // How to get free contiguous space ?
  free_space_ = MAX_MEMORY_BYTES;
}

inline bool check_cublas_error() {
  hipblasStatus_t status = cublasGetError();
  return status != HIPBLAS_STATUS_SUCCESS;
}

__device__ inline float square(float a) {
  return a * a;
}
inline void __getLastCudaError(const char *errorMessage, const char *file, const int line) {
 hipError_t err = hipGetLastError();
 if (hipSuccess != err) {
  fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString(err));
  exit(EXIT_FAILURE);
 }
}

void _Scale(cudamat* mat, float scale) {
  if (scale == 0) {
    hipMemset(mat->data_device, 0, sizeof(float) * mat->size[0] * mat->size[1]);
  } else if (scale != 1) {
    hipblasSscal(mat->size[0] * mat->size[1], scale, mat->data_device, 1);
  }
}

class AvgPooler {
 public:
  __device__ inline float operator()(const float a, const float b) const {
    return a + b;
  }
  __device__ inline float getBaseValue() const {
    return 0;
  }
  __device__ inline float output(const float a, const int regionSize) const {
    return a / regionSize;
  }
};

class MaxPooler {
 public:
  __device__ inline float operator()(const float a, const float b) const {
    return fmaxf(a, b);
  }
  __device__ inline float getBaseValue() const {
    return -2e38; 
  }
  __device__ inline float output(const float a, const int regionSize) const {
    return a;
  }
};

__global__ void kExpand(float *images, float* targets,
                        int num_images, int num_input_channels,
                        int image_size_y, int image_size_x,
                        int num_modules_y, int num_modules_x,
                        int kernel_size_y, int kernel_size_x,
                        int padding_y, int padding_x,
                        int stride_y, int stride_x,
                        int num_modules_batch, int module_id_offset) {
  int color = blockIdx.y;
  int src_module_id = module_id_offset + blockIdx.x;
  int dst_module_id = blockIdx.x;

  int module_id_x = src_module_id % num_modules_x;
  int module_id_y = src_module_id / num_modules_x;
  int startX = module_id_x * stride_x + padding_x;
  int startY = module_id_y * stride_y + padding_y;
  int Y, X;
  long target_id, source_id;
  images += num_images * image_size_x * image_size_y * color;
  targets += num_images * (dst_module_id + num_modules_batch * (kernel_size_y * kernel_size_x * color));
  for (int y = 0; y < kernel_size_y; y++) {
    Y = startY + y;
    for (int x = 0; x < kernel_size_x; x++) {
      X = startX + x;
      target_id = num_images * num_modules_batch * (x + kernel_size_x * y);
      source_id = num_images * (X + image_size_x * Y);
      if (X < 0 || X >= image_size_x || Y < 0 || Y >= image_size_y) {
        for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
          targets[target_id + im] = 0;
        }
      } else {
        for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
          targets[target_id + im] = images[source_id + im];
        }
      }
      __syncthreads();
    }
  }
}


__global__ void kOutpTraceMultiplyImages(float *expanded_images, float *expanded_derivs, 
                                         int num_images, int num_channels,
                                         int num_modules_batch, int kernel_size){
  int color = blockIdx.y;
  int module_id = blockIdx.x;

  expanded_images += num_images * num_modules_batch * kernel_size * color;
  expanded_images += num_images * module_id;
  expanded_derivs += num_images * num_modules_batch * color;
  expanded_derivs += num_images * module_id;

  for (int kpos = 0; kpos < kernel_size; kpos++) {
    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
      int image_idx = im + num_images * num_modules_batch * kpos;
      int deriv_idx = im;
      expanded_images[image_idx] *= expanded_derivs[deriv_idx];
    }
    __syncthreads();
  }

}


__global__ void kSetOnes(float *dest, int count){
  for (int i = threadIdx.x; i < count; i += blockDim.x) {
    dest[i] = 1;
  }
}

                                



template <class Pooler>
__global__ void kPool(float* images, float* targets,
                      int num_images, int num_input_channels,
                      int image_size_y, int image_size_x, int image_size_t,
                      int num_modules_y, int num_modules_x, int num_modules_t,
                      int kernel_size_y, int kernel_size_x, int kernel_size_t,
                      int padding_y, int padding_x, int padding_t,
                      int stride_y, int stride_x, int stride_t,
                      float scaleOutput, Pooler pooler) {
  const int color = blockIdx.y;
  const int num_colors = gridDim.y;
  const int num_modules = num_modules_y * num_modules_x * num_modules_t;

  long source_id, target_id;
  images  += num_images * image_size_x * image_size_y * color;
  targets += num_images * num_modules_x * num_modules_y * color;
  for (int module_id = blockIdx.x; module_id < num_modules; module_id += gridDim.x) {
    int module_id_x = module_id % num_modules_x;
    int module_id_y = (module_id / num_modules_x) % num_modules_y;
    int module_id_t = (module_id / num_modules_x) / num_modules_y;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    int startT = module_id_t * stride_t + padding_t;
    int endX = startX + kernel_size_x;
    int endY = startY + kernel_size_y;
    int endT = startT + kernel_size_t;
    target_id = num_images * (module_id_x + num_modules_x * (module_id_y + num_modules_y * num_colors * module_id_t));
    startX = MAX(startX, 0);
    startY = MAX(startY, 0);
    startT = MAX(startT, 0);
    endY   = MIN(endY  , image_size_y);
    endX   = MIN(endX  , image_size_x);
    endT   = MIN(endT  , image_size_t);
    int regionSize = (endX - startX) * (endY - startY) * (endT - startT);
    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
      float val = pooler.getBaseValue();
      for (int T = startT; T < endT; T++) {
        for (int Y = startY; Y < endY; Y++) {
          for (int X = startX; X < endX; X++) {
            source_id = num_images * (X + image_size_x * (Y + image_size_y * num_colors * T));
            val = pooler(val, images[source_id + im]);
          }
        }
      }
      targets[target_id + im] = scaleOutput * pooler.output(val, regionSize);
    }
  }
  __syncthreads();
}


__global__ void kAvgPoolUndo(float *derivs, float* targets,
                             int num_images, int num_input_channels,
                             int image_size_y, int image_size_x, int image_size_t,
                             int num_modules_y, int num_modules_x, int num_modules_t,
                             int kernel_size_y, int kernel_size_x, int kernel_size_t,
                             int padding_y, int padding_x, int padding_t,
                             int stride_y, int stride_x, int stride_t, float scaleOutput) {
  const int color = blockIdx.y;
  const int num_colors = gridDim.y;
  const int num_modules = num_modules_y * num_modules_x * num_modules_t;

  long source_id, target_id;
  derivs  += num_images * num_modules_x * num_modules_y * color;
  targets += num_images * image_size_x * image_size_y * color;
  for (int module_id = blockIdx.x; module_id < num_modules; module_id += gridDim.x) {
    int module_id_x = module_id % num_modules_x;
    int module_id_y = (module_id / num_modules_x) % num_modules_y;
    int module_id_t = (module_id / num_modules_x) / num_modules_y;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    int startT = module_id_t * stride_t + padding_t;
    int endX = startX + kernel_size_x;
    int endY = startY + kernel_size_y;
    int endT = startT + kernel_size_t;
    source_id = num_images * (module_id_x + num_modules_x * (module_id_y + num_modules_y * num_colors * module_id_t));
    startX = MAX(startX, 0);
    startY = MAX(startY, 0);
    startT = MAX(startT, 0);
    endY   = MIN(endY  , image_size_y);
    endX   = MIN(endX  , image_size_x);
    endT   = MIN(endT  , image_size_t);
    int regionSize = (endX - startX) * (endY - startY) * (endT - startT);

    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
      float val = scaleOutput * derivs[source_id + im] / regionSize;
      for (int T = startT; T < endT; T++) {
        for (int Y = startY; Y < endY; Y++) {
          for (int X = startX; X < endX; X++) {
            target_id = num_images * (X + image_size_x * (Y + image_size_y * num_colors * T)) + im;
            atomicAdd(&targets[target_id], val);
            __syncthreads();
          }
        }
      }
    }
  }
}


__global__ void kMaxPoolUndo(float * images, float *derivs, float* maxes, float* targets,
                        int num_images, int num_input_channels,
                        int image_size_y, int image_size_x, int image_size_t,
                        int num_modules_y, int num_modules_x, int num_modules_t,
                        int kernel_size_y, int kernel_size_x, int kernel_size_t,
                        int padding_y, int padding_x, int padding_t,
                        int stride_y, int stride_x, int stride_t, float scaleOutput) {
  const int color = blockIdx.y;
  const int num_colors = gridDim.y;
  const int num_modules = num_modules_y * num_modules_x * num_modules_t;

  long source_id, target_id;
  derivs  += num_images * num_modules_x * num_modules_y * color;
  maxes   += num_images * num_modules_x * num_modules_y * color;
  targets += num_images * image_size_x * image_size_y * color;
  images  += num_images * image_size_x * image_size_y * color;
  for (int module_id = blockIdx.x; module_id < num_modules; module_id += gridDim.x) {
    int module_id_x = module_id % num_modules_x;
    int module_id_y = (module_id / num_modules_x) % num_modules_y;
    int module_id_t = (module_id / num_modules_x) / num_modules_y;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    int startT = module_id_t * stride_t + padding_t;
    int endX = startX + kernel_size_x;
    int endY = startY + kernel_size_y;
    int endT = startT + kernel_size_t;
    source_id = num_images * (module_id_x + num_modules_x * (module_id_y + num_modules_y * num_colors * module_id_t));
    startX = MAX(startX, 0);
    startY = MAX(startY, 0);
    startT = MAX(startT, 0);
    endY   = MIN(endY  , image_size_y);
    endX   = MIN(endX  , image_size_x);
    endT   = MIN(endT  , image_size_t);
    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
      float val = scaleOutput * derivs[source_id + im];
      for (int T = startT; T < endT; T++) {
        for (int Y = startY; Y < endY; Y++) {
          for (int X = startX; X < endX; X++) {
            target_id = num_images * (X + image_size_x * (Y + image_size_y * num_colors * T)) + im;
            if (images[target_id] == maxes[source_id + im]) {
              atomicAdd(&targets[target_id], val);
            }
            __syncthreads();
          }
        }
      }
    }
  }
}

__global__ void kMaxPoolRprop(float * images, float *R_images, float* maxes, float* targets,
                              int num_images, int num_input_channels,
                              int image_size_y, int image_size_x,
                              int num_modules_y, int num_modules_x,
                              int kernel_size_y, int kernel_size_x,
                              int padding_y, int padding_x,
                              int stride_y, int stride_x, float scaleOutput) {
  int color = blockIdx.y;
  int num_modules = num_modules_y * num_modules_x;

  long input_id, output_id;
  R_images  += num_images * image_size_x * image_size_y * color;
  maxes  += num_images * num_modules * color;
  targets += num_images * num_modules * color;
  images += num_images * image_size_x * image_size_y * color;
  for (int module_id = blockIdx.x; module_id < num_modules; module_id += gridDim.x) {
    int module_id_x = module_id % num_modules_x;
    int module_id_y = module_id / num_modules_x;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    //output_id = num_images * module_id;
    int endY = startY + kernel_size_y;
    int endX = startX + kernel_size_x;
    startY = MAX(startY, 0);
    startX = MAX(startX, 0);
    endY   = MIN(endY  , image_size_y);
    endX   = MIN(endX  , image_size_x);
    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
      //float val = scaleOutput * derivs[source_id + im];
      for (int Y = startY; Y < endY; Y++) {
        for (int X = startX; X < endX; X++) {
          //target_id = num_images * (X + image_size_x * Y) + im;
          input_id = num_images * (X + image_size_x * Y) + im;
          output_id = num_images * module_id + im;
          if (images[input_id] == maxes[output_id]) {
            atomicAdd(&targets[output_id], R_images[input_id]);
          }
          __syncthreads();
        }
      }
    }
  }
}








__global__ void kContract(float *expanded_data, float* targets,
                          int num_images, int num_input_channels,
                          int image_size_y, int image_size_x,
                          int num_modules_y, int num_modules_x,
                          int kernel_size_y, int kernel_size_x,
                          int padding_y, int padding_x,
                          int stride_y, int stride_x,
                          int num_modules_batch, int module_id_offset) {
  int color = blockIdx.y;
  int dst_module_id = module_id_offset + blockIdx.x;
  int src_module_id = blockIdx.x;

  int module_id_x = dst_module_id % num_modules_x;
  int module_id_y = dst_module_id / num_modules_x;
  int startX = module_id_x * stride_x + padding_x;
  int startY = module_id_y * stride_y + padding_y;
  int Y, X;
  long target_id, source_id;
  targets += num_images * image_size_x * image_size_y * color;
  expanded_data  += num_images * (src_module_id + num_modules_batch * (kernel_size_y * kernel_size_x * color));
  for (int y = 0; y < kernel_size_y; y++) {
    Y = startY + y;
    for (int x = 0; x < kernel_size_x; x++) {
      X = startX + x;
      source_id = num_images * num_modules_batch * (x + kernel_size_x * y);
      target_id = num_images * (X + image_size_x * Y);
      if (X < 0 || X >= image_size_x || Y < 0 || Y >= image_size_y) {
        // do nothing.
      } else {
        for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
          atomicAdd(&targets[target_id + im], expanded_data[source_id + im]);
          __syncthreads();
        }
      }
    }
  }
}

__global__ void kWriteRows(float* data, float* target, int num_images,
                           int num_modules, int num_modules_batch,
                           int module_id_offset, float beta) {
  int c = blockIdx.y;
  int src_module_id = blockIdx.x;
  int dst_module_id = module_id_offset + blockIdx.x;

  data += num_images * (src_module_id + c * num_modules_batch);
  target += num_images * (dst_module_id + c * num_modules);

  for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
    target[im] = beta * data[im];
  }
}

__global__ void kReadRows(float* data, float* target,
                          int num_images, int num_modules,
                          int num_modules_batch, int module_id_offset) {
  int c = blockIdx.y;
  int src_module_id = module_id_offset + blockIdx.x;
  int dst_module_id = blockIdx.x;

  data += num_images * (src_module_id + c * num_modules);
  target += num_images * (dst_module_id + c * num_modules_batch);

  for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
    target[im] = data[im];
  }
}


__global__ void kWriteRowsMult(float* data, float* target,
                               int num_images, int num_modules,
                               int num_modules_batch, int module_id_offset,
                               float alpha, float beta) {
  int c = blockIdx.y;
  int src_module_id = blockIdx.x;
  int dst_module_id = module_id_offset + blockIdx.x;

  data += num_images * (src_module_id + c * num_modules_batch);
  target += num_images * (dst_module_id + c * num_modules);

  for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
    target[im] = alpha * target[im] + beta * data[im];
  }
}

__global__ void kCrossMapDenoms(float* data, float* denoms,
                                int num_locs, int batch_locs, int batch_offset,
                                float addScale, float powScale,
                                int num_filters, int k, bool blocked) {
  long loc_id = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  if (batch_offset + loc_id < num_locs) {
    data   += batch_offset + loc_id;
    denoms += loc_id;
    int prev_start = 0, prev_end = 0, start, end;
    float sum = 0;
    for (int j = 0; j < num_filters; j++) {
      start = blocked ? (j / k) * k : -k/2 + j;
      end = MIN(num_filters, start + k);
      start = MAX(0, start);
      for (int i = prev_start; i < start; i++) {
        sum -= square(data[i * num_locs]);
      }
      for (int i = prev_end; i < end; i++) {
        sum += square(data[i * num_locs]);
      }
      denoms[j * batch_locs] = __powf(1 + addScale * sum, -powScale - 1);
      prev_start = start;
      prev_end = end;
    }
  }
}

__global__ void kCrossMapRNorm(float* data, float* target,
                               int num_locs, float addScale, float powScale,
                               int num_filters, int k, bool blocked) {
  long loc_id = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  if (loc_id < num_locs) {
    data   += loc_id;
    target += loc_id;
    float sum = 0;
    int prev_start = 0, prev_end = 0, start, end;
    for (int j = 0; j < num_filters; j++) {
      start = blocked ? (j / k) * k : -k/2 + j;
      end = MIN(num_filters, start + k);
      start = MAX(0, start);
      for (int i = prev_start; i < start; i++) {
        sum -= square(data[i * num_locs]);
      }
      for (int i = prev_end; i < end; i++) {
        sum += square(data[i * num_locs]);
      }
      target[j * num_locs] = data[j * num_locs] * __powf(1 + addScale * sum, -powScale);
      prev_start = start;
      prev_end = end;
    }
  }
}

__global__ void kCrossMapRNormRprop(float* data, float* R_data, float* target,
                                    int num_locs, float addScale, float powScale,
                                    int num_filters, int k, bool blocked) {
  long loc_id = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  data   += loc_id;
  R_data += loc_id;
  target += loc_id;
  if (loc_id < num_locs) {
    for (int j = 0; j < num_filters; j++) {
      float sum_x2 = 0;
      float sum_xdx = 0;
      int start = blocked ? (j / k) * k : -k/2 + j;
      int end = MIN(num_filters, start + k);
      start = MAX(0, start);
      for (int i = start; i < end; i++) {
        sum_x2 += data[i * num_locs] * data[i * num_locs];
        sum_xdx += data[i * num_locs] * R_data[i * num_locs];
      }
      float denom = 1 + addScale * sum_x2;
      target[j * num_locs] = R_data[j * num_locs] * __powf(denom, -powScale)
        - 2 * addScale * powScale * data[j * num_locs] * __powf(denom, -powScale-1) * sum_xdx;
    }
  }
}

__global__ void kCrossMapRNormUndo(float* data, float* deriv, float* denoms, float* target,
                                   int num_locs, int batch_locs, int batch_offset, float addScale, float powScale,
                                   int num_filters, int k, bool blocked) {
  long loc_id = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  if (batch_offset + loc_id < num_locs) {
    data   += batch_offset + loc_id;
    target += batch_offset + loc_id;
    deriv  += batch_offset + loc_id;
    denoms += loc_id;
    float sum = 0;
    int prev_start = 0, prev_end = 0, start, end;
    for (int j = 0; j < num_filters; j++) {
      start = blocked ? (j / k) * k : -k + k/2 + j + 1;
      end = MIN(num_filters, start + k);
      start = MAX(0, start);
      for (int i = prev_start; i < start; i++) {
        sum -= deriv[i * num_locs] * data[i * num_locs] * denoms[i * batch_locs];
      }
      for (int i = prev_end; i < end; i++) {
        sum += deriv[i * num_locs] * data[i * num_locs] * denoms[i * batch_locs];
      }
      target[j * num_locs] = deriv[j * num_locs] * __powf(denoms[j * batch_locs], powScale / (powScale + 1)) -
                             2 * addScale * powScale * data[j * num_locs] * sum;
      prev_start = start;
      prev_end = end;
    }
  }
}

void _convUpGemm(cudamat* images, cudamat* filters, cudamat* targets,
                Shape4D images_shape, Shape4D filters_shape,
                Shape4D targets_shape, ConvDesc conv_desc,
                float scaleTargets, float scaleOutput, bool conv) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int input_channel_begin  = conv_desc.input_channel_begin;
    int input_channel_end    = conv_desc.input_channel_end;
    int output_channel_begin = conv_desc.output_channel_begin;
    int output_channel_end   = conv_desc.output_channel_end;
    int num_groups           = conv_desc.num_groups;

    if (output_channel_end == 0) output_channel_end = num_output_channels;
    if (input_channel_end == 0) input_channel_end = num_input_channels;

    int num_output_channels2 = targets_shape.shape[3];
    int num_modules_y        = targets_shape.shape[2];
    int num_modules_x        = targets_shape.shape[1];
    int num_images           = targets_shape.shape[0];

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images2          = images_shape.shape[0];

    int num_input_channels3  = filters_shape.shape[3];
    int kernel_size_y2       = filters_shape.shape[2];
    int kernel_size_x2       = filters_shape.shape[1];
    int num_output_channels3 = filters_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
    int filterModuleMult     = conv ? 1 : num_modules;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (output_channel_end - output_channel_begin == num_output_channels3);
    assert (num_input_channels == num_input_channels2);
    assert (input_channel_end - input_channel_begin == num_input_channels3 / filterModuleMult);
    assert (num_images == images->size[0]);
    assert (num_images == targets->size[0]);
    assert (num_output_channels3 == filters->size[0]);
    assert (image_size_y * image_size_x * num_input_channels == images->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels == targets->size[1]);
    assert (kernel_size_y * kernel_size_x * num_input_channels3 * filterModuleMult == filters->size[1]);
    assert (kernel_size_y == kernel_size_y2);
    assert (kernel_size_x == kernel_size_x2);
    assert (num_input_channels % num_groups == 0);
    assert (num_groups == 1);
    assert (input_channel_begin  >= 0);
    assert (output_channel_begin >= 0);
    assert (input_channel_end    <= num_input_channels);
    assert (output_channel_end   <= num_output_channels);
    assert (input_channel_begin  <= input_channel_end);
    assert (output_channel_begin <= output_channel_end);
    num_input_channels = input_channel_end - input_channel_begin;
    num_output_channels = output_channel_end - output_channel_begin;
    assert(num_input_channels  > 0);
    assert(num_output_channels > 0);
    float* w = filters->data_device;
    float* images_data = images->data_device + input_channel_begin * image_size_y * image_size_x * num_images;
    float* targets_data = targets->data_device + output_channel_begin * num_modules * num_images;

    int input_size = kernel_size_y * kernel_size_x * num_input_channels;
    int num_threads_x = MIN(num_images, NUM_THREADS_PER_BLOCK);
    
    float *expanded_images = NULL, *expanded_target = NULL;
    int num_modules_batch;
    
    int input_memory_size  = num_images * input_size * sizeof(float);
    int output_memory_size = num_images * num_output_channels * sizeof(float);
    if (free_space_ == 0) EstimateFreeSpace();
    int max_batch_size = free_space_ / (input_memory_size + output_memory_size);
    max_batch_size = MIN(max_batch_size, num_modules / filterModuleMult);
    max_batch_size = MIN(max_batch_size, MAX_BLOCKS_X);
    //max_batch_size = MAX(max_batch_size, 1);
    //printf("Free space %ld max batch size %d\n", free_space, max_batch_size);

    hipError_t err1, err2;
    err1 = hipMalloc((void**)&expanded_images,  max_batch_size * input_memory_size);
    err2 = hipMalloc((void**)&expanded_target, max_batch_size * output_memory_size);
    if (hipSuccess != err1 || hipSuccess != err2) {
      printf("Could not allocate memory.\n");
      num_modules_batch = 1;
    } else {
      num_modules_batch = max_batch_size;
    }

    int num_iter = DIVUP(num_modules, num_modules_batch);

    int module_id_start = 0;
    for (int i = 0; i < num_iter; i++) {
      int this_num_modules_batch = MIN(num_modules_batch, num_modules - module_id_start);
      //printf("Step %d num_modules %d\n", i, this_num_modules_batch);

      dim3 threads(num_threads_x);
      dim3 blocks = dim3(this_num_modules_batch, input_channel_end - input_channel_begin);
      kExpand<<<blocks, threads>>>(images_data, expanded_images,
                                   num_images, num_input_channels,
                                   image_size_y, image_size_x,
                                   num_modules_y, num_modules_x,
                                   kernel_size_y, kernel_size_x,
                                   padding_y, padding_x,
                                   stride_y, stride_x,
                                   this_num_modules_batch, module_id_start);
      if (!conv) w += num_output_channels * input_size;
      hipblasSgemm('n', 't', 
                  num_images * this_num_modules_batch, num_output_channels,
                  input_size,
                  1, expanded_images, num_images * this_num_modules_batch,
                  w, num_output_channels,
                  0, expanded_target, num_images * this_num_modules_batch);

      dim3 blocks2 = dim3(this_num_modules_batch, num_output_channels);
      if (scaleTargets == 0) {
        kWriteRows<<<blocks2, threads>>>(expanded_target, targets_data,
                                         num_images, num_modules,
                                         this_num_modules_batch, module_id_start,
                                         scaleOutput);
      } else {
        kWriteRowsMult<<<blocks2, threads>>>(expanded_target, targets_data,
                                         num_images, num_modules,
                                         this_num_modules_batch, module_id_start,
                                         scaleTargets, scaleOutput);
      }
      module_id_start += this_num_modules_batch;
    }
    hipFree(expanded_images);
    hipFree(expanded_target);
    getLastCudaError("convUpGemm: kernel execution failed");
}

void _convDownGemm(cudamat* derivs, cudamat* filters, cudamat* targets,
                Shape4D derivs_shape, Shape4D filters_shape,
                Shape4D targets_shape, ConvDesc conv_desc,
                float scaleTargets, float scaleOutput, bool conv) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int input_channel_begin  = conv_desc.input_channel_begin;
    int input_channel_end    = conv_desc.input_channel_end;
    int output_channel_begin = conv_desc.output_channel_begin;
    int output_channel_end   = conv_desc.output_channel_end;
    int num_groups           = conv_desc.num_groups;
    if (output_channel_end == 0) output_channel_end = num_output_channels;
    if (input_channel_end == 0) input_channel_end = num_input_channels;

    int num_output_channels2 = derivs_shape.shape[3];
    int num_modules_y        = derivs_shape.shape[2];
    int num_modules_x        = derivs_shape.shape[1];
    int num_images           = derivs_shape.shape[0];

    int num_input_channels2  = targets_shape.shape[3];
    int image_size_y         = targets_shape.shape[2];
    int image_size_x         = targets_shape.shape[1];
    int num_images2          = targets_shape.shape[0];

    int num_input_channels3  = filters_shape.shape[3];
    int kernel_size_y2       = filters_shape.shape[2];
    int kernel_size_x2       = filters_shape.shape[1];
    int num_output_channels3 = filters_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
    int filterModuleMult     = conv ? 1 : num_modules;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (output_channel_end - output_channel_begin == num_output_channels3);
    assert (num_input_channels == num_input_channels2);
    assert (input_channel_end - input_channel_begin == num_input_channels3 / filterModuleMult);
    assert (num_images2 == targets->size[0]);
    assert (num_images == derivs->size[0]);
    assert (num_output_channels3 == filters->size[0]);
    assert (image_size_y * image_size_x * num_input_channels2 == targets->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels2 == derivs->size[1]);
    assert (kernel_size_y * kernel_size_x * num_input_channels3 * filterModuleMult == filters->size[1]);
    assert (kernel_size_y == kernel_size_y2);
    assert (kernel_size_x == kernel_size_x2);
    assert (num_input_channels % num_groups == 0);
    assert (num_groups == 1);
    assert (input_channel_begin  >= 0);
    assert (output_channel_begin >= 0);
    assert (input_channel_end    <= num_input_channels);
    assert (output_channel_end   <= num_output_channels);
    assert (input_channel_begin  <= input_channel_end);
    assert (output_channel_begin <= output_channel_end);
    num_input_channels = input_channel_end - input_channel_begin;
    num_output_channels = output_channel_end - output_channel_begin;
    assert(num_input_channels  > 0);
    assert(num_output_channels > 0);
    float* w = filters->data_device;
    float* derivs_data = derivs->data_device + output_channel_begin * num_modules * num_images;
    float* targets_data = targets->data_device + input_channel_begin * image_size_y * image_size_x * num_images;
    
    int input_size = kernel_size_y * kernel_size_x * num_input_channels;
    int num_threads_x = MIN(num_images, NUM_THREADS_PER_BLOCK);
    float *expanded_target = NULL, *expanded_derivs = NULL;
    int num_modules_batch;

    int input_memory_size  = num_images * input_size * sizeof(float);
    int output_memory_size = num_images * num_output_channels * sizeof(float);
    if (free_space_ == 0) EstimateFreeSpace();
    int max_batch_size = free_space_ / (input_memory_size + output_memory_size);
    max_batch_size = MIN(max_batch_size, num_modules / filterModuleMult);
    max_batch_size = MIN(max_batch_size, MAX_BLOCKS_X);
    max_batch_size = MAX(max_batch_size, 1);

    hipError_t err1, err2;
    err1 = hipMalloc((void**)&expanded_target,  max_batch_size * input_memory_size);
    err2 = hipMalloc((void**)&expanded_derivs, max_batch_size * output_memory_size);
    if (hipSuccess != err1 || hipSuccess != err2) {
      printf("Out of memory\n");
      /*
      if (hipSuccess == err1) hipFree(expanded_target);
      if (hipSuccess == err2) hipFree(expanded_derivs);
      err1 = hipMalloc((void**)&expanded_target,  input_memory_size);
      err2 = hipMalloc((void**)&expanded_derivs, output_memory_size);
      if (hipSuccess != err1 || hipSuccess != err2) {
        printf("Out of memory on GPU! %s \n", hipGetErrorString(err1));
        printf("Out of memory on GPU! %s \n", hipGetErrorString(err2));
      }
      */
      num_modules_batch = 1;
    } else {
      num_modules_batch = max_batch_size;
    }

    int num_iter = DIVUP(num_modules, num_modules_batch);
   
    _Scale(targets, scaleTargets); 

    int module_id_start = 0;
    for (int i = 0; i < num_iter; i++) {
      int this_num_modules_batch = MIN(num_modules_batch, num_modules - module_id_start);
      //printf("Step %d num_modules %d\n", i, this_num_modules_batch);

      dim3 blocks = dim3(this_num_modules_batch, num_output_channels);
      dim3 threads(num_threads_x);
      kReadRows<<<blocks, threads>>>(derivs_data, expanded_derivs,
                                     num_images, num_modules,
                                     this_num_modules_batch, module_id_start);
      if (!conv) w += num_output_channels * input_size;
      hipblasSgemm('n', 'n', 
                  num_images * this_num_modules_batch, kernel_size_x * kernel_size_y * num_input_channels,
                  num_output_channels,
                  scaleOutput, expanded_derivs, num_images * this_num_modules_batch,
                  w, num_output_channels,
                  0, expanded_target, num_images * this_num_modules_batch);

      if (check_cublas_error()) {
        printf("Error in dot or before it.\n");
      }
      dim3 blocks2 = dim3(this_num_modules_batch, num_input_channels);
      kContract<<<blocks2, threads>>>(expanded_target, targets_data,
                                   num_images, num_input_channels,
                                   image_size_y, image_size_x,
                                   num_modules_y, num_modules_x,
                                   kernel_size_y, kernel_size_x,
                                   padding_y, padding_x,
                                   stride_y, stride_x,
                                   this_num_modules_batch, module_id_start);
      module_id_start += this_num_modules_batch;
    }
    hipFree(expanded_derivs);
    hipFree(expanded_target);
    getLastCudaError("convDownGemm: kernel execution failed");
}

void _convOutpGemm(cudamat* images, cudamat* derivs, cudamat* targets,
              Shape4D images_shape, Shape4D derivs_shape, Shape4D targets_shape,
              ConvDesc conv_desc, float scaleTargets, float scaleOutput, bool conv) {

    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int input_channel_begin  = conv_desc.input_channel_begin;
    int input_channel_end    = conv_desc.input_channel_end;
    int output_channel_begin = conv_desc.output_channel_begin;
    int output_channel_end   = conv_desc.output_channel_end;
    int num_groups           = conv_desc.num_groups;
    if (output_channel_end == 0) output_channel_end = num_output_channels;
    if (input_channel_end == 0) input_channel_end = num_input_channels;

    int num_output_channels2 = derivs_shape.shape[3];
    int num_modules_y        = derivs_shape.shape[2];
    int num_modules_x        = derivs_shape.shape[1];
    int num_images           = derivs_shape.shape[0];

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images2          = images_shape.shape[0];

    int num_input_channels3Mult  = targets_shape.shape[3];
    int kernel_size_y2       = targets_shape.shape[2];
    int kernel_size_x2       = targets_shape.shape[1];
    int num_output_channels3 = targets_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
    int filterModuleMult     = conv ? 1 : num_modules;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (output_channel_end - output_channel_begin == num_output_channels3);
    assert (num_input_channels == num_input_channels2);
    assert (input_channel_end - input_channel_begin == num_input_channels3Mult / filterModuleMult);
    assert (num_images2 == images->size[0]);
    assert (num_images == derivs->size[0]);
    assert (num_output_channels3 == targets->size[0]);
    assert (image_size_y * image_size_x * num_input_channels2 == images->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels2 == derivs->size[1]);
    assert (kernel_size_y2 * kernel_size_x2 * num_input_channels3Mult == targets->size[1]);
    assert (kernel_size_y == kernel_size_y2);
    assert (kernel_size_x == kernel_size_x2);
    assert (num_input_channels % num_groups == 0);
    assert (num_groups == 1);
    assert (input_channel_begin  >= 0);
    assert (output_channel_begin >= 0);
    assert (input_channel_end    <= num_input_channels);
    assert (output_channel_end   <= num_output_channels);
    assert (input_channel_begin  <= input_channel_end);
    assert (output_channel_begin <= output_channel_end);
    if (output_channel_end == 0) output_channel_end = num_output_channels;
    if (input_channel_end == 0) input_channel_end = num_input_channels;
    num_input_channels = input_channel_end - input_channel_begin;
    num_output_channels = output_channel_end - output_channel_begin;
    assert(num_input_channels  > 0);
    assert(num_output_channels > 0);
    float* dw = targets->data_device;
    float* images_data = images->data_device + input_channel_begin * image_size_y * image_size_x * num_images;
    float* derivs_data = derivs->data_device + output_channel_begin * num_modules * num_images;
    
    int input_size = kernel_size_y * kernel_size_x * num_input_channels;
    int num_threads_x = MIN(num_images, NUM_THREADS_PER_BLOCK);
    
    float *expanded_images = NULL, *expanded_derivs = NULL;
    int num_modules_batch;

    int input_memory_size  = num_images * input_size * sizeof(float);
    int output_memory_size = num_images * num_output_channels * sizeof(float);
    if (free_space_ == 0) EstimateFreeSpace();
    int max_batch_size = free_space_ / (input_memory_size + output_memory_size);
    max_batch_size = MIN(max_batch_size, num_modules / filterModuleMult);
    max_batch_size = MIN(max_batch_size, MAX_BLOCKS_X);
    max_batch_size = MAX(max_batch_size, 1);

    hipError_t err1, err2;
    err1 = hipMalloc((void**)&expanded_images,  max_batch_size * input_memory_size);
    err2 = hipMalloc((void**)&expanded_derivs, max_batch_size * output_memory_size);
    if (hipSuccess != err1 || hipSuccess != err2) {
      printf("Out of memory.\n");
      num_modules_batch = 1;
    } else {
      num_modules_batch = max_batch_size;
    }

    int num_iter = DIVUP(num_modules, num_modules_batch);

    _Scale(targets, scaleTargets);

    int module_id_start = 0;
    dim3 threads(num_threads_x);
    for (int i = 0; i < num_iter; i++) {
      int this_num_modules_batch = MIN(num_modules_batch, num_modules - module_id_start);
      //printf("Step %d num_modules %d\n", i, this_num_modules_batch);

      dim3 blocks = dim3(this_num_modules_batch, num_output_channels);
      kReadRows<<<blocks, threads>>>(derivs_data, expanded_derivs,
                                     num_images, num_modules,
                                     this_num_modules_batch, module_id_start);
      dim3 blocks2 = dim3(this_num_modules_batch, num_input_channels);
      kExpand<<<blocks2, threads>>>(images_data, expanded_images,
                                   num_images, num_input_channels,
                                   image_size_y, image_size_x,
                                   num_modules_y, num_modules_x,
                                   kernel_size_y, kernel_size_x,
                                   padding_y, padding_x,
                                   stride_y, stride_x,
                                   this_num_modules_batch, module_id_start);
      if (!conv) dw += num_output_channels * input_size;
      hipblasSgemm('t', 'n', 
                  num_output_channels,
                  input_size,
                  num_images * this_num_modules_batch,
                  scaleOutput, expanded_derivs, num_images * this_num_modules_batch,
                  expanded_images, num_images * this_num_modules_batch,
                  1, dw, num_output_channels);
      if (check_cublas_error()) {
        printf("Error in dot or before it.\n");
      }
      module_id_start += this_num_modules_batch;
    }
    hipFree(expanded_derivs);
    hipFree(expanded_images);
    getLastCudaError("convOutpGemm: kernel execution failed");
}

void _convCovarianceGemm(cudamat* images, cudamat* y1_targets, cudamat* y2_targets,
                         Shape4D images_shape, Shape4D y2_targets_shape,
                         ConvDesc conv_desc, float scaleTargets, float scaleOutput) {

    int num_input_channels   = conv_desc.num_input_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int input_channel_begin  = conv_desc.input_channel_begin;
    int input_channel_end    = conv_desc.input_channel_end;
    int num_groups           = conv_desc.num_groups;
    if (input_channel_end == 0) input_channel_end = num_input_channels;

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images           = images_shape.shape[0];

    int num_modules_y        = image_size_y;
    int num_modules_x        = image_size_x;
    int num_modules          = num_modules_y * num_modules_x;

    int num_input_channels3  = y2_targets_shape.shape[3];
    int kernel_size_xy       = y2_targets_shape.shape[2];
    int kernel_size_xy2      = y2_targets_shape.shape[1];
    int num_input_channels4  = y2_targets_shape.shape[0];

  
    // Consistency checks.
    assert (num_input_channels == num_input_channels2);
    assert (input_channel_end - input_channel_begin == num_input_channels3);
    assert (num_input_channels3 == num_input_channels4);
    assert (kernel_size_xy == kernel_size_x * kernel_size_y);
    assert (kernel_size_xy2 == kernel_size_xy);
    assert (image_size_y * image_size_x * num_input_channels2 == images->size[1]);
    assert (kernel_size_xy * num_input_channels3 == y2_targets->size[1]);
    assert (kernel_size_xy * num_input_channels3 == y1_targets->size[0] * y1_targets->size[1]);
    assert (num_input_channels % num_groups == 0);
    assert (num_groups == 1);
    assert (input_channel_begin  >= 0);
    assert (input_channel_end    <= num_input_channels);
    assert (input_channel_begin  <= input_channel_end);
    if (input_channel_end == 0) input_channel_end = num_input_channels;
    num_input_channels = input_channel_end - input_channel_begin;
    assert(num_input_channels  > 0);
    float* Sigma = y2_targets->data_device;
    float* mu = y1_targets->data_device;
    float* images_data = images->data_device + input_channel_begin * image_size_y * image_size_x * num_images;
    
    int input_size = kernel_size_y * kernel_size_x * num_input_channels;
    int num_threads_x = MIN(num_images, NUM_THREADS_PER_BLOCK);
    
    float *expanded_images = NULL, *ones = NULL;
    int num_modules_batch;

    int input_memory_size  = num_images * input_size * sizeof(float);
    if (free_space_ == 0) EstimateFreeSpace();
    int max_batch_size = free_space_ / input_memory_size;
    max_batch_size = MIN(max_batch_size, num_modules);
    max_batch_size = MIN(max_batch_size, MAX_BLOCKS_X);
    max_batch_size = MAX(max_batch_size, 1);

    int num_ones = num_images * max_batch_size;
    int ones_size = num_ones * sizeof(float);
    

    hipError_t err1, err2;
    //printf("A\n");
    //getLastCudaError("convCovarianceGemm: kernel execution failed [A]");
    err1 = hipMalloc((void**)&expanded_images,  max_batch_size * input_memory_size);
    err2 = hipMalloc((void**)&ones, ones_size);
    if (hipSuccess != err1 || hipSuccess != err2) {
      printf("Out of memory.\n");
      num_modules_batch = 1;
    } else {
      num_modules_batch = max_batch_size;
    }

    kSetOnes<<<1, 512>>>(ones, num_ones);

    int num_iter = DIVUP(num_modules, num_modules_batch);

    _Scale(y1_targets, scaleTargets);
    _Scale(y2_targets, scaleTargets);

    //printf("B\n");
    //getLastCudaError("convCovarianceGemm: kernel execution failed [B]");

    int module_id_start = 0;
    dim3 threads(num_threads_x);
    for (int i = 0; i < num_iter; i++) {
      int this_num_modules_batch = MIN(num_modules_batch, num_modules - module_id_start);
      //printf("Step %d num_modules %d\n", i, this_num_modules_batch);

      dim3 blocks2 = dim3(this_num_modules_batch, num_input_channels);
      kExpand<<<blocks2, threads>>>(images_data, expanded_images,
                                   num_images, num_input_channels,
                                   image_size_y, image_size_x,
                                   num_modules_y, num_modules_x,
                                   kernel_size_y, kernel_size_x,
                                   padding_y, padding_x,
                                   stride_y, stride_x,
                                   this_num_modules_batch, module_id_start);

      //printf("C\n");
      //getLastCudaError("convCovarianceGemm: kernel execution failed [C]");
      hipblasSgemm('t', 'n', 
                  input_size,
                  input_size,
                  num_images * this_num_modules_batch,
                  scaleOutput, expanded_images, num_images * this_num_modules_batch,
                  expanded_images, num_images * this_num_modules_batch,
                  1, Sigma, input_size);

      if (check_cublas_error()) {
        printf("Error in sgemm or before it.\n");
      }

      //printf("D\n");
      //getLastCudaError("convCovarianceGemm: kernel execution failed [D]");
      hipblasSgemv('t',
                  num_images * this_num_modules_batch,
                  input_size,
                  scaleOutput, expanded_images, num_images * this_num_modules_batch,
                  ones, 1, 1, mu, 1);
      //printf("E\n");
      
      if (check_cublas_error()) {
        printf("Error in sgemv or before it.\n");
      }
      module_id_start += this_num_modules_batch;
    }
    //getLastCudaError("convCovarianceGemm: kernel execution failed (before freeing ones)");
    hipFree(ones);
    //getLastCudaError("convCovarianceGemm: kernel execution failed (after freeing ones)");
    hipFree(expanded_images);
    getLastCudaError("convCovarianceGemm: kernel execution failed (final)");
}


void _convCovariance2Gemm(cudamat* images, cudamat* images2, cudamat* targets,
                          Shape4D images_shape, Shape4D images2_shape, Shape4D targets_shape,
                          ConvDesc conv_desc, float scaleTargets, float scaleOutput) {

    int num_input_channels   = conv_desc.num_input_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int input_channel_begin  = conv_desc.input_channel_begin;
    int input_channel_end    = conv_desc.input_channel_end;
    int num_groups           = conv_desc.num_groups;
    if (input_channel_end == 0) input_channel_end = num_input_channels;

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images           = images_shape.shape[0];

    int num_modules_y        = image_size_y;
    int num_modules_x        = image_size_x;
    int num_modules          = num_modules_y * num_modules_x;

    int num_input_channels3  = targets_shape.shape[3];
    int kernel_size_xy       = targets_shape.shape[2];
    int kernel_size_xy2      = targets_shape.shape[1];
    int num_input_channels4  = targets_shape.shape[0];

    
  
    // Consistency checks.
    assert(images2_shape.shape[0] == images_shape.shape[0]);
    assert(images2_shape.shape[1] == images_shape.shape[1]);
    assert(images2_shape.shape[2] == images_shape.shape[2]);
    assert(images2_shape.shape[3] == images_shape.shape[3]);
    assert (num_input_channels == num_input_channels2);
    assert (input_channel_end - input_channel_begin == num_input_channels3);
    assert (input_channel_end - input_channel_begin == num_input_channels4);
    assert (kernel_size_xy == kernel_size_x * kernel_size_y);
    assert (kernel_size_xy2 == kernel_size_xy);
    assert (image_size_y * image_size_x * num_input_channels2 == images->size[1]);
    assert (kernel_size_xy * num_input_channels3 == targets->size[1]);
    assert (num_input_channels % num_groups == 0);
    assert (num_groups == 1);
    assert (input_channel_begin  >= 0);
    assert (input_channel_end    <= num_input_channels);
    assert (input_channel_begin  <= input_channel_end);
    if (input_channel_end == 0) input_channel_end = num_input_channels;
    num_input_channels = input_channel_end - input_channel_begin;
    assert(num_input_channels  > 0);
    float* Sigma = targets->data_device;
    float* images_data = images->data_device + input_channel_begin * image_size_y * image_size_x * num_images;
    float* images2_data = images2->data_device + input_channel_begin * image_size_y * image_size_x * num_images;
    
    int input_size = kernel_size_y * kernel_size_x * num_input_channels;
    int num_threads_x = MIN(num_images, NUM_THREADS_PER_BLOCK);
    
    float *expanded_images = NULL, *expanded_images2 = NULL;
    int num_modules_batch;

    int input_memory_size  = num_images * input_size * sizeof(float);
    if (free_space_ == 0) EstimateFreeSpace();
    int max_batch_size = free_space_ / (2 * input_memory_size);
    max_batch_size = MIN(max_batch_size, num_modules);
    max_batch_size = MIN(max_batch_size, MAX_BLOCKS_X);
    max_batch_size = MAX(max_batch_size, 1);

    hipError_t err1, err2;
    err1 = hipMalloc((void**)&expanded_images,  max_batch_size * input_memory_size);
    err2 = hipMalloc((void**)&expanded_images2, max_batch_size * input_memory_size);
    if (hipSuccess != err1) {
      printf("Out of memory.\n");
      num_modules_batch = 1;
    } else {
      num_modules_batch = max_batch_size;
    }

    int num_iter = DIVUP(num_modules, num_modules_batch);

    _Scale(targets, scaleTargets);

    int module_id_start = 0;
    dim3 threads(num_threads_x);
    for (int i = 0; i < num_iter; i++) {
      int this_num_modules_batch = MIN(num_modules_batch, num_modules - module_id_start);
      //printf("Step %d num_modules %d\n", i, this_num_modules_batch);

      dim3 blocks2 = dim3(this_num_modules_batch, num_input_channels);
      kExpand<<<blocks2, threads>>>(images_data, expanded_images,
                                   num_images, num_input_channels,
                                   image_size_y, image_size_x,
                                   num_modules_y, num_modules_x,
                                   kernel_size_y, kernel_size_x,
                                   padding_y, padding_x,
                                   stride_y, stride_x,
                                   this_num_modules_batch, module_id_start);
      kExpand<<<blocks2, threads>>>(images2_data, expanded_images2,
                                   num_images, num_input_channels,
                                   image_size_y, image_size_x,
                                   num_modules_y, num_modules_x,
                                   kernel_size_y, kernel_size_x,
                                   padding_y, padding_x,
                                   stride_y, stride_x,
                                   this_num_modules_batch, module_id_start);
      
      hipblasSgemm('t', 'n', 
                  input_size,
                  input_size,
                  num_images * this_num_modules_batch,
                  scaleOutput, expanded_images, num_images * this_num_modules_batch,
                  expanded_images2, num_images * this_num_modules_batch,
                  1, Sigma, input_size);
      if (check_cublas_error()) {
        printf("Error in dot or before it.\n");
      }
      module_id_start += this_num_modules_batch;
    }
    hipFree(expanded_images);
    hipFree(expanded_images2);
    getLastCudaError("convCovarianceGemm: kernel execution failed");
}


void _convInnerpGemm(cudamat* images, cudamat* derivs, cudamat* targets,
                        Shape4D images_shape, Shape4D derivs_shape, Shape4D targets_shape,
                        ConvDesc conv_desc, float scaleTargets, float scaleOutput, bool conv) {

    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int input_channel_begin  = conv_desc.input_channel_begin;
    int input_channel_end    = conv_desc.input_channel_end;
    int output_channel_begin = conv_desc.output_channel_begin;
    int output_channel_end   = conv_desc.output_channel_end;
    int num_groups           = conv_desc.num_groups;
    if (output_channel_end == 0) output_channel_end = num_output_channels;
    if (input_channel_end == 0) input_channel_end = num_input_channels;

    int num_output_channels2 = derivs_shape.shape[3];
    int num_modules_y        = derivs_shape.shape[2];
    int num_modules_x        = derivs_shape.shape[1];
    int num_images           = derivs_shape.shape[0];

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images2          = images_shape.shape[0];

    assert (targets_shape.shape[3] == 1);
    int kernel_size_y2       = targets_shape.shape[2];
    int kernel_size_x2       = targets_shape.shape[1];
    assert (targets_shape.shape[0] == 1);

    int num_modules          = num_modules_y * num_modules_x;
    int filterModuleMult     = conv ? 1 : num_modules;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    //assert (output_channel_end - output_channel_begin == num_output_channels3);
    assert (num_input_channels == num_input_channels2);
    //assert (input_channel_end - input_channel_begin == num_input_channels3Mult / filterModuleMult);
    assert (num_images2 == images->size[0]);
    assert (num_images == derivs->size[0]);
    //assert (num_output_channels3 == targets->size[0]);
    assert (image_size_y * image_size_x * num_input_channels2 == images->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels2 == derivs->size[1]);
    //assert (kernel_size_y2 * kernel_size_x2 * num_input_channels3Mult == targets->size[1]);
    assert (kernel_size_y == kernel_size_y2);
    assert (kernel_size_x == kernel_size_x2);
    assert (num_input_channels % num_groups == 0);
    assert (num_groups == 1);
    assert (input_channel_begin  >= 0);
    assert (output_channel_begin >= 0);
    assert (input_channel_end    <= num_input_channels);
    assert (output_channel_end   <= num_output_channels);
    assert (input_channel_begin  <= input_channel_end);
    assert (output_channel_begin <= output_channel_end);
    if (output_channel_end == 0) output_channel_end = num_output_channels;
    if (input_channel_end == 0) input_channel_end = num_input_channels;
    num_input_channels = input_channel_end - input_channel_begin;
    num_output_channels = output_channel_end - output_channel_begin;
    assert(num_input_channels  > 0);
    assert(num_output_channels > 0);
    float* dw = targets->data_device;
    float* images_data = images->data_device + input_channel_begin * image_size_y * image_size_x * num_images;
    float* derivs_data = derivs->data_device + output_channel_begin * num_modules * num_images;

    // RBG
    assert (num_input_channels == num_output_channels);

    int input_size = kernel_size_y * kernel_size_x * num_input_channels;
    int num_threads_x = MIN(num_images, NUM_THREADS_PER_BLOCK);
    
    float *expanded_images = NULL, *expanded_derivs = NULL, *expanded_images_sum = NULL, *ones = NULL;
    int num_modules_batch;

    int input_memory_size  = num_images * input_size * sizeof(float);
    int output_memory_size = num_images * num_output_channels * sizeof(float);
    if (free_space_ == 0) EstimateFreeSpace();
    int max_batch_size = free_space_ / (input_memory_size + output_memory_size);
    max_batch_size = MIN(max_batch_size, num_modules / filterModuleMult);
    max_batch_size = MIN(max_batch_size, MAX_BLOCKS_X);
    max_batch_size = MAX(max_batch_size, 1);

    int expanded_images_sum_size = kernel_size_x * kernel_size_y * num_input_channels * sizeof(float);
    int num_ones = max(num_images * max_batch_size, num_input_channels);
    int ones_size = num_ones * sizeof(float);

    hipError_t err1, err2, err3, err4;
    err1 = hipMalloc((void**)&expanded_images,  max_batch_size * input_memory_size);
    err2 = hipMalloc((void**)&expanded_derivs, max_batch_size * output_memory_size);
    err3 = hipMalloc((void**)&expanded_images_sum, expanded_images_sum_size);
    err4 = hipMalloc((void**)&ones, ones_size);
    if (hipSuccess != err1 || hipSuccess != err2 || hipSuccess != err3 || hipSuccess != err4) {
      printf("Out of memory.\n");
      num_modules_batch = 1;
    } else {
      num_modules_batch = max_batch_size;
    }

    kSetOnes<<<1, 512>>>(ones, num_ones);

    int num_iter = DIVUP(num_modules, num_modules_batch);

    _Scale(targets, scaleTargets);

    int module_id_start = 0;
    dim3 threads(num_threads_x);
    for (int i = 0; i < num_iter; i++) {
      int this_num_modules_batch = MIN(num_modules_batch, num_modules - module_id_start);
      //printf("Step %d num_modules %d\n", i, this_num_modules_batch);

      dim3 blocks = dim3(this_num_modules_batch, num_output_channels);
      kReadRows<<<blocks, threads>>>(derivs_data, expanded_derivs,
                                     num_images, num_modules,
                                     this_num_modules_batch, module_id_start);
      dim3 blocks2 = dim3(this_num_modules_batch, num_input_channels);
      
      kExpand<<<blocks2, threads>>>(images_data, expanded_images,
                                    num_images, num_input_channels,
                                    image_size_y, image_size_x,
                                    num_modules_y, num_modules_x,
                                    kernel_size_y, kernel_size_x,
                                    padding_y, padding_x,
                                    stride_y, stride_x,
                                    this_num_modules_batch, module_id_start);
      
      if (!conv) dw += num_output_channels * input_size;

      kOutpTraceMultiplyImages<<<blocks2, threads>>>(expanded_images, expanded_derivs,
                                                     num_images, num_input_channels,
                                                     this_num_modules_batch, kernel_size_x * kernel_size_y);

      // sum over images and modules by summing columns
      int num_rows = num_images * this_num_modules_batch;
      int num_cols = kernel_size_x * kernel_size_y * num_input_channels;
      hipblasSgemv('t', num_rows, num_cols, 1,
                  expanded_images, num_rows,
                  ones, 1,
                  0, expanded_images_sum, 1);

      // sum over channels by summing rows
      num_rows = kernel_size_x * kernel_size_y;
      num_cols = num_input_channels;
      hipblasSgemv('n', num_rows, num_cols, scaleOutput,
                  expanded_images_sum, num_rows,
                  ones, 1,
                  1, dw, 1);

      
      if (check_cublas_error()) {
        printf("Error in dot or before it.\n");
      }
      module_id_start += this_num_modules_batch;
    }
    hipFree(expanded_derivs);
    hipFree(expanded_images);
    hipFree(expanded_images_sum);
    hipFree(ones);
    getLastCudaError("convOutpGemm: kernel execution failed");
}

template <class Pooler>
void _convPoolGemm(cudamat* images, cudamat* targets,
                Shape4D images_shape, Shape4D targets_shape,
                ConvDesc conv_desc, float scaleTargets, float scaleOutput, Pooler pooler) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int kernel_size_t        = conv_desc.kernel_size_t;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int stride_t             = conv_desc.stride_t;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int padding_t            = conv_desc.padding_t;

    int num_output_channels2 = targets_shape.shape[3];
    int num_modules_y        = targets_shape.shape[2];
    int num_modules_x        = targets_shape.shape[1];
    int num_images           = targets_shape.shape[0];

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images2          = images_shape.shape[0];

    int image_size_t         = num_input_channels2 / num_input_channels;
    int num_modules_t        = num_output_channels2 / num_output_channels;
    int num_modules          = num_modules_y * num_modules_x * num_modules_t;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_input_channels2 % image_size_t == 0);
    assert (num_input_channels == num_input_channels2 / image_size_t);
    assert (num_output_channels2 % num_modules_t == 0);
    assert (num_output_channels == num_output_channels2 / num_modules_t);
    assert (num_images == images->size[0]);
    assert (num_images == targets->size[0]);
    assert (image_size_t * image_size_y * image_size_x * num_input_channels == images->size[1]);
    assert (num_modules_t * num_modules_y * num_modules_x * num_output_channels == targets->size[1]);

    _Scale(targets, scaleTargets);

    dim3 threads(NUM_THREADS_PER_BLOCK);
    int num_blocks_x = MIN(MAX_BLOCKS_X, num_modules);
    dim3 blocks = dim3(num_blocks_x, num_input_channels);
    kPool<<<blocks, threads>>>(images->data_device, targets->data_device,
                               num_images, num_input_channels,
                               image_size_y, image_size_x, image_size_t,
                               num_modules_y, num_modules_x, num_modules_t,
                               kernel_size_y, kernel_size_x, kernel_size_t,
                               padding_y, padding_x, padding_t,
                               stride_y, stride_x, stride_t, scaleOutput,
                               pooler);
    getLastCudaError("convLocalPool: kernel execution failed");
}

void _avgPoolUndoGemm(cudamat* derivs, cudamat* targets,
                Shape4D derivs_shape, Shape4D targets_shape,
                ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
 
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int kernel_size_t        = conv_desc.kernel_size_t;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int stride_t             = conv_desc.stride_t;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int padding_t            = conv_desc.padding_t;

    int num_output_channels2 = derivs_shape.shape[3];
    int num_modules_y        = derivs_shape.shape[2];
    int num_modules_x        = derivs_shape.shape[1];
    int num_images           = derivs_shape.shape[0];

    int num_input_channels2  = targets_shape.shape[3];
    int image_size_y         = targets_shape.shape[2];
    int image_size_x         = targets_shape.shape[1];
    int num_images2          = targets_shape.shape[0];

    int image_size_t         = num_input_channels2 / num_input_channels;
    int num_modules_t        = num_output_channels2 / num_output_channels;
    int num_modules          = num_modules_y * num_modules_x * num_modules_t;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_input_channels2 % image_size_t == 0);
    assert (num_input_channels == num_input_channels2 / image_size_t);
    assert (num_output_channels2 % num_modules_t == 0);
    assert (num_output_channels == num_output_channels2 / num_modules_t);
    assert (num_images == targets->size[0]);
    assert (num_images == derivs->size[0]);
    assert (image_size_t * image_size_y * image_size_x * num_input_channels == targets->size[1]);
    assert (num_modules_t * num_modules_y * num_modules_x * num_output_channels == derivs->size[1]);

    _Scale(targets, scaleTargets);

    dim3 threads(NUM_THREADS_PER_BLOCK);
    int num_blocks_x = MIN(MAX_BLOCKS_X, num_modules);
    dim3 blocks = dim3(num_blocks_x, num_input_channels);
    kAvgPoolUndo<<<blocks, threads>>>(derivs->data_device, targets->data_device,
                               num_images, num_input_channels,
                               image_size_y, image_size_x, image_size_t,
                               num_modules_y, num_modules_x, num_modules_t,
                               kernel_size_y, kernel_size_x, kernel_size_t,
                               padding_y, padding_x, padding_t,
                               stride_y, stride_x, stride_t, scaleOutput);
    getLastCudaError("avgPoolUndo: kernel execution failed");
}

void _maxPoolUndoGemm(cudamat* images, cudamat* derivs, cudamat* maxes,
                      cudamat* targets, Shape4D targets_shape,
                      Shape4D derivs_shape, ConvDesc conv_desc,
                      float scaleTargets, float scaleOutput) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int kernel_size_t        = conv_desc.kernel_size_t;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int stride_t             = conv_desc.stride_t;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int padding_t            = conv_desc.padding_t;

    int num_output_channels2 = derivs_shape.shape[3];
    int num_modules_y        = derivs_shape.shape[2];
    int num_modules_x        = derivs_shape.shape[1];
    int num_images           = derivs_shape.shape[0];

    int num_input_channels2  = targets_shape.shape[3];
    int image_size_y         = targets_shape.shape[2];
    int image_size_x         = targets_shape.shape[1];
    int num_images2          = targets_shape.shape[0];

    int image_size_t         = num_input_channels2 / num_input_channels;
    int num_modules_t        = num_output_channels2 / num_output_channels;
    int num_modules          = num_modules_y * num_modules_x * num_modules_t;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_input_channels2 % image_size_t == 0);
    assert (num_input_channels == num_input_channels2 / image_size_t);
    assert (num_output_channels2 % num_modules_t == 0);
    assert (num_output_channels == num_output_channels2 / num_modules_t);
    assert (num_images == targets->size[0]);
    assert (num_images == derivs->size[0]);
    assert (image_size_t * image_size_y * image_size_x * num_input_channels == targets->size[1]);
    assert (num_modules_t * num_modules_y * num_modules_x * num_output_channels == derivs->size[1]);


    _Scale(targets, scaleTargets);

    dim3 threads(NUM_THREADS_PER_BLOCK);
    int num_blocks_x = MIN(MAX_BLOCKS_X, num_modules);
    dim3 blocks = dim3(num_blocks_x, num_input_channels);
    kMaxPoolUndo<<<blocks, threads>>>(images->data_device, derivs->data_device,
                               maxes->data_device, targets->data_device,
                               num_images, num_input_channels,
                               image_size_y, image_size_x, image_size_t,
                               num_modules_y, num_modules_x, num_modules_t,
                               kernel_size_y, kernel_size_x, kernel_size_t,
                               padding_y, padding_x, padding_t,
                               stride_y, stride_x, stride_t, scaleOutput);
    getLastCudaError("maxPoolUndo: kernel execution failed");
}

void _maxPoolRpropGemm(cudamat* images, cudamat* R_images, cudamat* maxes, cudamat* targets,
                Shape4D images_shape, Shape4D maxes_shape, 
                ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;

    int num_output_channels2 = maxes_shape.shape[3];
    int num_modules_y        = maxes_shape.shape[2];
    int num_modules_x        = maxes_shape.shape[1];
    int num_images           = maxes_shape.shape[0];

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images2          = images_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (num_input_channels == num_input_channels2);
    assert (num_images == R_images->size[0]);
    assert (num_images == targets->size[0]);
    assert (num_modules_y * num_modules_x * num_input_channels == targets->size[1]);
    assert (image_size_y * image_size_x * num_output_channels == R_images->size[1]);

    if (scaleTargets == 0) {
      hipMemset(targets->data_device, 0, sizeof(float) * targets->size[0] * targets->size[1]);
    } else if (scaleTargets != 1) {
      hipblasSscal(sizeof(float) * targets->size[0] * targets->size[1], scaleTargets, targets->data_device, 1);
    }

    dim3 threads(128);
    int num_blocks_x = MIN(4096, num_modules);
    dim3 blocks = dim3(num_blocks_x, num_input_channels);
    kMaxPoolRprop<<<blocks, threads>>>(images->data_device, R_images->data_device,
                               maxes->data_device, targets->data_device,
                               num_images, num_input_channels,
                               image_size_y, image_size_x,
                               num_modules_y, num_modules_x,
                               kernel_size_y, kernel_size_x,
                               padding_y, padding_x,
                               stride_y, stride_x, scaleOutput);
    getLastCudaError("avgPoolUndo: kernel execution failed");
}



void _CrossMapRNorm(cudamat* images, cudamat* targets, int num_filters, int sizeF, float addScale, float powScale, bool blocked) {
  int num_locs = (images->size[0] * images->size[1]) / num_filters;
  int num_blocks = DIVUP(num_locs, NUM_THREADS_PER_BLOCK);
  kCrossMapRNorm<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(images->data_device, targets->data_device,
                 num_locs, addScale, powScale, num_filters, sizeF, blocked);
  getLastCudaError("_CrossMapRNorm: kernel execution failed");
}

void _CrossMapRNormUndo(cudamat* outGrads, cudamat* images, cudamat* targets,
                        int num_filters, int sizeF, float addScale,
                        float powScale, bool blocked) {
  int num_locs = (images->size[0] * images->size[1]) / num_filters;
  int batch_offset = 0;

  float *denoms;
  if (free_space_ == 0) EstimateFreeSpace();
  int max_batch_size = free_space_ / (sizeof(float) * num_filters);
  max_batch_size = MIN(num_locs, max_batch_size);
  max_batch_size = MIN(num_locs, MAX_BLOCKS_X);
  hipError_t err;
  err = hipMalloc((void**)&denoms, max_batch_size * num_filters * sizeof(float));
  if (hipSuccess != err) {
    printf("Out of memory on GPU!\n");
  }
  int num_batches = DIVUP(num_locs, max_batch_size);
  for (int i = 0; i < num_batches; i++) {
    int batch_size = MIN(max_batch_size, num_locs - batch_offset);
    int num_blocks = DIVUP(batch_size, NUM_THREADS_PER_BLOCK);
    kCrossMapDenoms<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(images->data_device, denoms, num_locs, batch_size,
                    batch_offset, addScale, powScale, num_filters, sizeF, blocked);

    kCrossMapRNormUndo<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(images->data_device, outGrads->data_device, denoms,
                       targets->data_device, num_locs, batch_size, batch_offset,
                       addScale, powScale, num_filters, sizeF, blocked);
    batch_offset += batch_size;
  }

  hipFree(denoms);
  getLastCudaError("_CrossMapRNormUndo: kernel execution failed");
}

void _CrossMapRNormRprop(cudamat* images, cudamat* R_images, cudamat* targets, int num_filters, int sizeF, float addScale, float powScale, bool blocked) {
  int num_locs = (images->size[0] * images->size[1]) / num_filters;
  int threads = 512;
  int num_blocks = DIVUP(num_locs, threads);
  kCrossMapRNormRprop<<<num_blocks, threads>>>(images->data_device, R_images->data_device, targets->data_device,
                                               num_locs, addScale, powScale, num_filters, sizeF, blocked);
  getLastCudaError("_CrossMapRNormRprop: kernel execution failed");
}

#ifdef __cplusplus
extern "C" {
#endif

void convUpGemm(cudamat* images, cudamat* filters, cudamat* targets,
                Shape4D* images_shape, Shape4D* filters_shape,
                Shape4D* targets_shape, ConvDesc conv_desc,
                float scaleTargets) {
  /*
  printf("image shape %d %d %d %d\n", images_shape->shape[0], images_shape->shape[1], images_shape->shape[2], images_shape->shape[3]);
  printf("filters shape %d %d %d %d\n", filters_shape->shape[0], filters_shape->shape[1], filters_shape->shape[2], filters_shape->shape[3]);
  printf("targets shape %d %d %d %d\n", targets_shape->shape[0], targets_shape->shape[1], targets_shape->shape[2], targets_shape->shape[3]);
  printf("Convolution : kernel_size_y %d kernel_size_x %d stride_y %d stride_x %d padding_y %d padding_x %d num_input_channels %d num_output_channels %d num_groups %d\n",
          conv_desc.kernel_size_y, conv_desc.kernel_size_x, conv_desc.stride_x, conv_desc.stride_y, conv_desc.padding_y, conv_desc.padding_x, conv_desc.num_input_channels,
          conv_desc.num_output_channels, conv_desc.num_groups);
          */
  _convUpGemm(images, filters, targets, *images_shape, *filters_shape,
              *targets_shape, conv_desc, scaleTargets, 1.0, true);
}
 
void convDownGemm(cudamat* derivs, cudamat* filters, cudamat* targets,
              Shape4D* derivs_shape, Shape4D* filters_shape,
              Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets) {
  _convDownGemm(derivs, filters, targets, *derivs_shape, *filters_shape,
                *targets_shape, conv_desc, scaleTargets, 1.0, true);
}

void convOutpGemm(cudamat* images, cudamat* derivs, cudamat* targets,
              Shape4D* images_shape, Shape4D* derivs_shape, Shape4D* targets_shape,
              ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
  _convOutpGemm(images, derivs, targets, *images_shape, *derivs_shape,
              *targets_shape, conv_desc, scaleTargets, scaleOutput, true);
}

void convCovarianceGemm(cudamat* images, cudamat* y1_targets, cudamat* y2_targets,
                        Shape4D* images_shape, Shape4D* y2_targets_shape,
                        ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
  _convCovarianceGemm(images, y1_targets, y2_targets, *images_shape, *y2_targets_shape, conv_desc,
                      scaleTargets, scaleOutput);
}

void convCovariance2Gemm(cudamat* images, cudamat* images2, cudamat* targets,
                         Shape4D* images_shape, Shape4D* images2_shape, Shape4D* targets_shape,
                         ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
  _convCovariance2Gemm(images, images2, targets, *images_shape, *images2_shape, *targets_shape, conv_desc,
                       scaleTargets, scaleOutput);
}
  
void convInnerpGemm(cudamat* images, cudamat* derivs, cudamat* targets,
                    Shape4D* images_shape, Shape4D* derivs_shape, Shape4D* targets_shape,
                    ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
  _convInnerpGemm(images, derivs, targets, *images_shape, *derivs_shape,
                  *targets_shape, conv_desc, scaleTargets, scaleOutput, true);
}

void localUpGemm(cudamat* images, cudamat* filters, cudamat* targets,
                Shape4D* images_shape, Shape4D* filters_shape,
                Shape4D* targets_shape, ConvDesc conv_desc,
                float scaleTargets) {
  _convUpGemm(images, filters, targets, *images_shape, *filters_shape,
              *targets_shape, conv_desc, scaleTargets, 1.0, false);
}
void localDownGemm(cudamat* derivs, cudamat* filters, cudamat* targets,
              Shape4D* derivs_shape, Shape4D* filters_shape,
              Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets) {
  _convDownGemm(derivs, filters, targets, *derivs_shape, *filters_shape,
                *targets_shape, conv_desc, scaleTargets, 1.0, false);
}

void localOutpGemm(cudamat* images, cudamat* derivs, cudamat* targets,
              Shape4D* images_shape, Shape4D* derivs_shape, Shape4D* targets_shape,
              ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
  _convOutpGemm(images, derivs, targets, *images_shape, *derivs_shape,
              *targets_shape, conv_desc, scaleTargets, scaleOutput, false);
}

void MaxPoolGemm(cudamat* images, cudamat* targets, Shape4D* images_shape,
             Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets, float scaleOutput){
  MaxPooler pooler;
  _convPoolGemm<MaxPooler>(images, targets, *images_shape, *targets_shape,
                           conv_desc, scaleTargets, scaleOutput, pooler);
}

void AvgPoolGemm(cudamat* images, cudamat* targets, Shape4D* images_shape,
             Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets, float scaleOutput){
  AvgPooler pooler;
  _convPoolGemm<AvgPooler>(images, targets, *images_shape, *targets_shape,
                           conv_desc, scaleTargets, scaleOutput, pooler);
}

void MaxPoolUndoGemm(cudamat* images, cudamat* maxGrads, cudamat* maxActs,
                 cudamat* targets, Shape4D* images_shape, Shape4D* maxGrads_shape,
                 ConvDesc conv_desc, float scaleTargets) {
  _maxPoolUndoGemm(images, maxGrads, maxActs, targets, *images_shape,
                   *maxGrads_shape, conv_desc, scaleTargets, 1);
}

void MaxPoolRpropGemm(cudamat* images, cudamat* R_images, cudamat* maxActs,
                      cudamat* targets, Shape4D* images_shape, Shape4D* maxGrads_shape,
                      ConvDesc conv_desc, float scaleTargets) {
  _maxPoolRpropGemm(images, R_images, maxActs, targets, *images_shape,
                    *maxGrads_shape, conv_desc, scaleTargets, 1);
}

void AvgPoolUndoGemm(cudamat* avgGrads, cudamat* targets, Shape4D* avgGrads_shape,
                 Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets) {
  _avgPoolUndoGemm(avgGrads, targets, *avgGrads_shape, *targets_shape, conv_desc,
                   scaleTargets, 1);
}

void UpSampleGemm(cudamat* images, cudamat* targets, Shape4D* images_shape,
              Shape4D* targets_shape, int factor, float scaleTargets) { 
  ConvDesc conv_desc;
  conv_desc.kernel_size_y = factor;
  conv_desc.kernel_size_x = factor;
  conv_desc.stride_y = factor;
  conv_desc.stride_x = factor;
  conv_desc.padding_y = 0;
  conv_desc.padding_x = 0;
  conv_desc.num_input_channels = images_shape->shape[3];
  conv_desc.num_output_channels = targets_shape->shape[3];
  conv_desc.output_channel_end = targets_shape->shape[3];
  conv_desc.input_channel_end = images_shape->shape[3];
  conv_desc.input_channel_begin = 0;
  conv_desc.output_channel_begin = 0;
  conv_desc.num_groups = 1;
  _avgPoolUndoGemm(images, targets, *images_shape, *targets_shape, conv_desc,
                   scaleTargets, factor * factor);
}

void DownSampleGemm(cudamat* images, cudamat* targets, Shape4D* images_shape, Shape4D* targets_shape, int factor) {
  AvgPooler pooler = AvgPooler();
  ConvDesc conv_desc;
  conv_desc.kernel_size_y = factor;
  conv_desc.kernel_size_x = factor;
  conv_desc.stride_y = factor;
  conv_desc.stride_x = factor;
  conv_desc.padding_y = 0;
  conv_desc.padding_x = 0;
  conv_desc.num_input_channels = images_shape->shape[3];
  conv_desc.num_output_channels = targets_shape->shape[3];
  conv_desc.output_channel_end = targets_shape->shape[3];
  conv_desc.input_channel_end = images_shape->shape[3];
  conv_desc.input_channel_begin = 0;
  conv_desc.output_channel_begin = 0;
  conv_desc.num_groups = 1;
  _convPoolGemm<AvgPooler>(images, targets, *images_shape, *targets_shape,
                           conv_desc, 0, 1, pooler);
}

void ResponseNormCrossMapGemm(
  cudamat* images, cudamat* targets, int num_filters, int sizeF, float addScale,
  float powScale, bool blocked) {
  _CrossMapRNorm(images, targets, num_filters, sizeF, addScale, powScale, blocked);
}

void ResponseNormCrossMapUndoGemm(
  cudamat* outGrads, cudamat* inputs, cudamat* targets, int num_filters,
  int sizeF, float addScale, float powScale, bool blocked) {
  _CrossMapRNormUndo(outGrads, inputs, targets, num_filters, sizeF, addScale,
                     powScale, blocked);
}

void ResponseNormCrossMapRpropGemm(
  cudamat* images, cudamat* R_images, cudamat* targets, int num_filters, int sizeF, float addScale,
  float powScale, bool blocked) {
  _CrossMapRNormRprop(images, R_images, targets, num_filters, sizeF, addScale, powScale, blocked);
}
  
void Scale(cudamat* mat, float scale) {
  _Scale(mat, scale);
}

#ifdef __cplusplus
}
#endif
